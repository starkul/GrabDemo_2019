#include "hip/hip_runtime.h"
#include <stdio.h>  
#include "hip/hip_runtime.h"  
#include ""  

#include <iostream>
#include "hip/hip_runtime.h"  
#include ""  
#include "hip/device_functions.h"
//#include "sm_20_atomic_functions.h"
using namespace std;
// �������
__global__ void TwoPoint_Correction(unsigned short *gpu_img, int Length, int Width, float *pTP_Gain, float *pTP_Bias)
{
	for (int i = 0; i < 768; i++)
		for (int j = 0; j < 1024; j++)
			gpu_img[j + i * 1024] = unsigned short(pTP_Gain[j + i * 1024] * gpu_img[j + i * 1024] + pTP_Bias[j + i * 1024]);

}

// ä�����
__global__ void Blind_On_Correction(unsigned short *gpu_img, int Length, int Width, unsigned short *pBlind_Ram)
{
	float mean = 0; int count = 0;
	for (int i = 0; i < 768; i++)
		for (int j = 0; j < 1024; j++)
			if (pBlind_Ram[j + i * 1024] == 1) //��ΪäԪ���򽫸õ��Ϊ��Χ���ƽ��
			{
				mean = 0; count = 0;//��Ե��������
				if (i > 0)
				{
					count++;
					mean += gpu_img[j + (i-1) * 1024];
				}
				if (i < 768)
				{
					count++;
					mean += gpu_img[j + (i + 1) * 1024];
				}
				if (j > 0)
				{
					count++;
					mean += gpu_img[(j - 1) + i * 1024];
				}
				if (j < 1024)
				{
					count++;
					mean += gpu_img[(j + 1) + i * 1024];
				}
				gpu_img[j + i * 1024] = unsigned short(mean/count);
			}

}
//---------ֱ��ͼ����----------
__global__ void Histogram_Enhancement(unsigned short *gpu_img)
{
		//----------------------���������------------------------
		unsigned short Histogram_Count[65536] = { 0 };   //ֱ��ͼ��ǿ��
		float pHistogram_Enhancement[65536] = {};

		for (int i = 0; i < 768; i++)
			for (int j = 0; j < 1024; j++)
			{
				Histogram_Count[gpu_img[j + i * 1024]] = Histogram_Count[gpu_img[j + i * 1024]] + 1;
				//int k = Histogram_Count[gpu_img[j + i * 1024]];
				//k = 1;
			}

		//------------
		float sum = 0;

		//-----------����ֱ��ͼ��--------------
		for (int i = 0; i < 65536; i++)
		{
			sum = sum + Histogram_Count[i];
			pHistogram_Enhancement[i] = sum / 768 / 1024;
		}


		for (int i = 0; i < 768; i++)
			for (int j = 0; j < 1024; j++)
			{
				unsigned short k = gpu_img[j + i * 1024];
				gpu_img[j + i * 1024] =  65535 * pHistogram_Enhancement[k];
				//gpu_img[j + i * 1024] =  //pHistogram[0]*gpu_img[j + i * 1024];
				 //gpu_img[j + i * 1024] =  pHistogram[30001]*65535;
				//int k = pHistogram[gpu_img[j * 1024]];
			}
	
	//for (int i = 0; i < 768; i++)
	//	for (int j = 0; j < 1024; j++)
	//	{
	//		gpu_img[j + i * 1024] = 0;
	//	}

}

// ��д��ҪGPU����Ĺ��ܺ���
extern "C"
hipError_t Image_Solution(unsigned short *Image,int Width,int Length,float *pTP_Gain, float *pTP_Bias,int TP_On,int Blind_On, unsigned short *pBlind_Ram,int Histogram_On)  //  pTP_Gain,pTP_Biasָ�������������----TP_On��ʾ�Ƿ����������
{
	int size = 1; //���ú��߳���
	unsigned short *dev_img = 0;
	float *dev_pTP_Gain = 0, *dev_pTP_Bias = 0;
	unsigned short *dev_pBlind_Ram = 0;
	hipError_t cudaStatus;

	//size = pHistogram[65535];
	//size = pHistogram[20000];
	//size = pHistogram[10000];
	// Choose which GPU to run on, change this on a multi-GPU system.  
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//---------------------------ͼ���ڴ濪��---------------------------------------------------
	// ���ٴ��ͼ����ڴ�    .  
	cudaStatus = hipMalloc((void**)&dev_img, Length * Width * sizeof(unsigned short));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// ��ͼ���ڴ�浽CUDA�ڴ���  
	cudaStatus = hipMemcpy(dev_img, Image, Length * Width * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.  
	//�˴�Ϊ���� GPU�Ĳ��������������ַ
	////�˺����ĵ��ã�ע��<<<1,1>>>����һ��1�������̸߳���ֻ��һ���߳̿飻�ڶ���1������һ���߳̿���ֻ��һ���̡߳�
	//Solution_Kernel << <1, size >> >(dev_img, Length, Width, pTP_Gain, pTP_Bias);

	//----------------------------��������н������ڴ濪��---------------------------------------
	if (TP_On > 0)
	{
		cudaStatus = hipMalloc((void**)&dev_pTP_Gain, Length * Width * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Gain hipMalloc failed!");
			goto Error;
		}
		// ��ͼ���ڴ�浽CUDA�ڴ���  
		cudaStatus = hipMemcpy(dev_pTP_Gain, pTP_Gain, Length * Width * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Gain hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_pTP_Bias, Length * Width * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Bias hipMalloc failed!");
			goto Error;
		}
		// ��ͼ���ڴ�浽CUDA�ڴ���  
		cudaStatus = hipMemcpy(dev_pTP_Bias, pTP_Gain, Length * Width * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Bias hipMemcpy failed!");
			goto Error;
		}
		TwoPoint_Correction << <1, size >> >(dev_img, Length, Width, dev_pTP_Gain, dev_pTP_Bias);
	}

	//____________________________________________________________________________

	//--------------------------------äԪ����ʵ��--------------------------------
	if (Blind_On > 0)
	{
		cudaStatus = hipMalloc((void**)&dev_pBlind_Ram, Length * Width * sizeof(unsigned short));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Gain hipMalloc failed!");
			goto Error;
		}
		// ��ͼ���ڴ�浽CUDA�ڴ���  
		cudaStatus = hipMemcpy(dev_pBlind_Ram, pBlind_Ram, Length * Width * sizeof(unsigned short), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "dev_pTP_Gain hipMemcpy failed!");
			goto Error;
		}

		Blind_On_Correction << <1, size >> >(dev_img, Length, Width, dev_pBlind_Ram);
	}

	//--------------------------------ֱ��ͼ��ǿʵ��--------------------------------
	if (Histogram_On > 0)
	{
		Histogram_Enhancement << <1, 1 >> >(dev_img);
	}

	//---------------------------------------------------------------------------------------
	// Check for any errors launching the kernel  
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns  
	// any errors encountered during the launch.  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.  
	cudaStatus = hipMemcpy(Image, dev_img, Length * Width * sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//Ӧ��ÿ�ζ���Ҫ����ڴ��
	//hipFree(dev_img);
Error:
	hipFree(dev_img);
	hipFree(dev_pTP_Gain); hipFree(dev_pTP_Bias);
	hipFree(dev_pBlind_Ram);
	return cudaStatus;
}

//------------------------------------------------------------------------------
//                      GPU ��������
//------------------------------------------------------------------------------

//----------------------ֱ��ͼ��ǿ----------------------------
__global__ void Create_Histogram1(unsigned short *gpu_img, unsigned int * dev_Histogram)
{
	//ͳ��ֱ��ͼ
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//dev_Histogram[gpu_img[i]] += 1;
	unsigned short value =  gpu_img[gpu_img[i]];  //  ??? ͳ��ֱ��ͼֻ��Ҫȡ��ǰ�ĻҶ�ֵ�ɣ�
	//unsigned short value = gpu_img[i];
	//ԭ�Ӳ�������������
	atomicAdd(&dev_Histogram[value], 1);
}

__global__ void Clear_Histogram(unsigned int* dev_Histogram)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	dev_Histogram[i] = 0;
}

__global__ void Create_Histogram2(unsigned short* dev_img, unsigned int* dev_Histogram, int Height, int Width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned short count = dev_Histogram_Float[dev_img[i]];
	dev_img[i] = double(dev_Histogram[dev_img[i]]) / Height / Width * 65535;

	//double tmp = double(dev_Histogram[dev_img[i]]) / 768 / 1024;
	//dev_img[i] =  tmp * 65535 ;

	//dev_img[i] = unsigned short(dev_Histogram[dev_img[i]] / 16 * 65535 / 49152 + dev_Histogram[dev_img[i]] % 16);
	//dev_img[i] = unsigned short(dev_Histogram[dev_img[i]] / Height * 65535 / Width + dev_Histogram[dev_img[i]] % Height * 65535 / Width);
}
__global__ void Create_Histogram3(unsigned short *gpu_img, float* dev_Histogram_float)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//gpu_img[i] = dev_Histogram[gpu_img[i]] / 1024 / 768 ;
	gpu_img[i] =  65535 * dev_Histogram_float[gpu_img[i]];
	//gpu_img[i] = 0;
}

//------------------------��������------------------------
__global__ void Linear_1(unsigned short *gpu_img, unsigned short* dev_Histogram)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (gpu_img[i] > dev_Histogram[0])  //���ֵ�洢
		dev_Histogram[0] = gpu_img[i];
	if (gpu_img[i] < dev_Histogram[1])  //��Сֵ�洢
		dev_Histogram[1] = gpu_img[i];


		
/*
//����û����
for (int i=0;i<1024*768;i++)
if (gpu_img[i] > dev_Histogram[0])  //���ֵ�洢
dev_Histogram[0] = gpu_img[i];
*/

}
__global__ void Find_Max(unsigned short *g_idata, unsigned short *g_odata, unsigned short *gate_index)
{
	__shared__ int sdata[1024];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned int i = blockIdx.x * gate_index[1] + threadIdx.x;

	if(tid < *gate_index)
		sdata[tid] = g_idata[i];
	else 
		sdata[tid] = 0;

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			if (sdata[tid] < sdata[tid + s])
				sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}
__global__ void Find_Min(unsigned short *g_idata, unsigned short *g_odata, unsigned short *gate_index)
{
	__shared__ int sdata[1024];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned int i = blockIdx.x * 768 + threadIdx.x;

	if (tid < *gate_index)
		sdata[tid] = g_idata[i];
	else
		sdata[tid] = 65535;

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			if (sdata[tid] > sdata[tid + s])
				sdata[tid] = sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}
__global__ void Linear_2(unsigned short *gpu_img, unsigned short* dev_Histogram)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	gpu_img[i] = unsigned short (float(65535 / (dev_Histogram[0] - dev_Histogram[1]))*(gpu_img[i] - dev_Histogram[1]));  //��������
	//gpu_img[i] = unsigned short(65535 *(gpu_img[i] - dev_Histogram[1]) / (dev_Histogram[0] - dev_Histogram[1]));  //��������
}
//--------------------------------ֱ��ͼ��ǿʵ��--------------------------------
extern "C"
hipError_t GPU_Histogram_Enhancement(unsigned short *Image, unsigned int *Histogram,float *Histogram_Float,unsigned short *dev_img, unsigned int* dev_Histogram, float* dev_Histogram_Float, int Height, int Width)  //  pTP_Gain,pTP_Biasָ�������������----TP_On��ʾ�Ƿ����������
{

	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dev_img, Image, Height * Width * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}
	unsigned short *dev_index = 0;
/*	cudaStatus = hipMalloc((void**)&dev_index, 2 * sizeof(unsigned short));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "p_index hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(dev_index, &Width, 2 * sizeof(unsigned short), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_index+1, &Height, 2 * sizeof(unsigned short), hipMemcpyHostToDevice);
*/
	unsigned int tmp[65536];
	unsigned int *ptmp = &tmp[0];
	unsigned short tmp_short[300000];
	unsigned short tmp_float[65536];
	tmp[0] = 0;

/*	//����ͼ��Image�Ƿ��롣
	for (int i = 0; i < Height * Width; i++)
	{
		if (tmp[0] < Image[i])
			tmp[0] = Image[i];
		if (tmp[1] > Image[i])
			tmp[1] = Image[i];
	}
*/
/*	// ����ͼ���Ƿ���GPU
	hipMemcpy(Image, dev_img, 2 * sizeof(unsigned short), hipMemcpyDeviceToHost);
	*/

	//--------------------------------ֱ��ͼ��ǿʵ��--------------------------------
	//dim3 threadsPerBlock(1024);
	//dim3 numBlocks(768);
	dim3 threadsPerBlock(Width);
	dim3 numBlocks(Height);
	// ֱ��ͼ����

	Clear_Histogram << <256, 256 >> > (dev_Histogram);
	cudaStatus = hipMemcpy(ptmp, dev_Histogram, 65536 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(tmp_short, dev_img, 65536 * sizeof(unsigned short), hipMemcpyDeviceToHost);

	Create_Histogram1 << <numBlocks, threadsPerBlock >> > (dev_img, dev_Histogram);  //1.ͳ�Ƹ����Ҷ�ֵ������
	cudaStatus = hipMemcpy(ptmp, dev_Histogram, 65536 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(Histogram, dev_Histogram, 65536 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	long tValue = Height * Width;
	//Histogram[0]�Ѿ���ֵ��
	//�۲�ptmp���ݣ����ͳ���Ƿ���ȷ��
	for (int i = 1; i < 65536; i++)
	{
		Histogram[i] = Histogram[i - 1] + Histogram[i];
//		tmp_float[i] = Histogram[i] / tValue;
		tmp[i] = Histogram[i];
	}
	cudaStatus = hipMemcpy(dev_Histogram, Histogram, 65536 * sizeof(unsigned int), hipMemcpyHostToDevice);
	//��Ҫ���ݷֱ��ʵ�����ǿ����
	Create_Histogram2 << <numBlocks, threadsPerBlock >> > (dev_img, dev_Histogram, Height, Width);
	cudaStatus = hipMemcpy(tmp_short, dev_img,300000 * sizeof(unsigned short), hipMemcpyDeviceToHost);
/*	Histogram_Float[0] = Histogram[0] / tValue;
	for (int i = 1; i < 65536; i++)
	{
		Histogram_Float[i] = Histogram_Float[i - 1] + Histogram[i] / tValue;
		tmp_float[i] = Histogram_Float[i];
	}

	cudaStatus = hipMemcpy(dev_Histogram_Float, Histogram_Float, 65536 * sizeof(float), hipMemcpyHostToDevice);
*/
	unsigned short max = 0;
	for (int i = 0; i < 300000; i++)
		if (max < tmp_short[i])
			max = tmp_short[i];

	//hipMemcpy(ptmp, dev_Histogram, 65536 * sizeof(long), hipMemcpyDeviceToHost);
	//Create_Histogram3 << <numBlocks, threadsPerBlock >> > (dev_img, dev_Histogram_float);
	//Histogram_Enhancement << <1, 1 >> >(dev_img);  //����

	//-------------��������------------------

	//�����ֵ��Сֵ
/*	hipMemcpy(dev_Histogram, ptmp, 2 * sizeof(unsigned short), hipMemcpyHostToDevice); //���±�ֵ
	//Linear_1 << <numBlocks, threadsPerBlock >> > (dev_img, dev_Histogram);
	Find_Max <<< Height, Width >>> (dev_img, dev_Histogram,&dev_index[0]);  //0 --- width  \\  1------height
	Find_Max <<< 1, 1024 >>> (dev_Histogram, dev_Histogram, &dev_index[1]);
	hipMemcpy(ptmp, dev_Histogram, 1 * sizeof(unsigned short), hipMemcpyDeviceToHost);
	Find_Min << < Height, Width >> > (dev_img, dev_Histogram,&dev_index[0]);
	Find_Min << < 1, 1024 >> > (dev_Histogram, dev_Histogram,&dev_index[1]);
	hipMemcpy(ptmp+1, dev_Histogram+1, 1 * sizeof(unsigned short), hipMemcpyDeviceToHost);
	hipMemcpy(dev_Histogram, ptmp, 2 * sizeof(unsigned short), hipMemcpyHostToDevice); //���±�ֵ

	Linear_2 << <numBlocks, threadsPerBlock >> > (dev_img, dev_Histogram);
	*/
	//---------------------------------------------------------------------------------------
	// Check for any errors launching the kernel  
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns  
	// any errors encountered during the launch.  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.  
	cudaStatus = hipMemcpy(Image, dev_img, Height * Width * sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	//Ӧ��ÿ�ζ���Ҫ����ڴ��
	//hipFree(dev_img);

	return cudaStatus;
}

//*****************************************************************************************

//----------------------�������������-------------------------
__global__ void GPU_TwoPoint_Helper(unsigned short *gpu_img, float* dev_pTP_Gain, float* dev_pTP_Bias)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	gpu_img[i] = unsigned short(dev_pTP_Gain[i] * gpu_img[i] + dev_pTP_Bias[i]);
}
//-----------------------��������ӿں���--------------------------
extern "C"
hipError_t GPU_TwoPoint_Correction(unsigned short *Image, unsigned short *dev_img, float* dev_pTP_Gain, float* dev_pTP_Bias, int Height, int Width)  //  pTP_Gain,pTP_Biasָ�������������----TP_On��ʾ�Ƿ����������
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dev_img, Image, Height * Width * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}
	//--------------------------------�����������--------------------------------
	dim3 threadsPerBlock(Width);
	dim3 numBlocks(Height);
	GPU_TwoPoint_Helper << <numBlocks, threadsPerBlock >> > (dev_img, dev_pTP_Gain, dev_pTP_Bias);
	//TwoPoint_Correction << <1, 1 >> >(dev_img, Height, Width, dev_pTP_Gain, dev_pTP_Bias);
	//---------------------------------------------------------------------------------------
	// Check for any errors launching the kernel  
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns  
	// any errors encountered during the launch.  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.  
	cudaStatus = hipMemcpy(Image, dev_img, Height * Width * sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	//Ӧ��ÿ�ζ���Ҫ����ڴ��
	//hipFree(dev_img);

	return cudaStatus;
}

//*****************************************************************************************

//----------------------äԪ����������-------------------------
__global__ void GPU_Blind__Helper(unsigned short *gpu_img, unsigned short* dev_pBlind_Ram, int Height, int Width)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	if (dev_pBlind_Ram[i*blockDim.x+j] == 1) //����1��ʾΪäԪ
	{
		//��äԪ�滻Ϊ��Χ�ĸ����ص�ƽ��
		unsigned short n = 0; double sum = 0;
		if (j > 0)
		{
			sum += gpu_img[i*blockDim.x + j - 1]; n++;
		}
		if (j < Width)
		{
			sum += gpu_img[i*blockDim.x + j + 1]; n++;
		}
		if (i > 0)
		{
			sum += gpu_img[(i-1)*blockDim.x + j + 1]; n++;
		}
		if (i < Height)
		{
			sum += gpu_img[(i+1)*blockDim.x + j + 1]; n++;
		}
		if (n != 0)
			gpu_img[i*blockDim.x + j] = sum / n;
	}
}
//-----------------------äԪ����--------------------------
extern "C"
hipError_t GPU_Blind_Correction(unsigned short *Image, unsigned short *dev_img, unsigned short *dev_pBlind_Ram, int Height, int Width)  //  pTP_Gain,pTP_Biasָ�������������----TP_On��ʾ�Ƿ����������
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dev_img, Image, Height * Width * sizeof(unsigned short), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}
	//--------------------------------�����������--------------------------------
	dim3 threadsPerBlock(Width);
	dim3 numBlocks(Height);
	GPU_Blind__Helper << <numBlocks, threadsPerBlock >> > (dev_img, dev_pBlind_Ram, Height, Width);
	//Blind_On_Correction << <1, 1 >> > (dev_img, Height, Width, dev_pBlind_Ram);
	
/*	// ���Դ���
	unsigned short *ptmp = new unsigned short[768*1024];
	hipMemcpy(ptmp, dev_pBlind_Ram, 768 * 1024 * sizeof(unsigned short), hipMemcpyDeviceToHost);
	unsigned short kk = ptmp[700 * 1024 + 980];
	kk = ptmp[700 * 1024 + 981];
*/
	//---------------------------------------------------------------------------------------
	// Check for any errors launching the kernel  
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns  
	// any errors encountered during the launch.  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.  
	cudaStatus = hipMemcpy(Image, dev_img, Height * Width * sizeof(unsigned short), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	//Ӧ��ÿ�ζ���Ҫ����ڴ��
	//hipFree(dev_img);

	return cudaStatus;
}

