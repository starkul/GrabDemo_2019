#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hip/device_functions.h"

__constant__ float color_weight[4 * 256];
__constant__ float space_weight[1024];

USHORT *dImage = NULL;   //original image
USHORT *dTemp = NULL;   //temp array for iterations
size_t pitch;

texture<uchar4, 2, hipReadModeElementType> rgbaTex;//�����������ϵ


__device__ float colorLenGaussian(uchar4 a, uchar4 b)
{
	//����ﵽ����Ч������ע�͵�sqrt,ʹ��ɫ������
	USHORT mod = (USHORT)sqrt(((float)b.x - (float)a.x) * ((float)b.x - (float)a.x) +
		((float)b.y - (float)a.y) * ((float)b.y - (float)a.y) +
		((float)b.z - (float)a.z) * ((float)b.z - (float)a.z) +
		((float)b.w - (float)a.w) * ((float)b.w - (float)a.w));

	return color_weight[mod];
}
__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(fabs(rgba.y));
	rgba.z = __saturatef(fabs(rgba.z));
	rgba.w = __saturatef(fabs(rgba.w));
	return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}
__device__ float4 rgbaIntToFloat(uint c)
{
	float4 rgba;
	rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
	rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;  //  /255.0f;
	rgba.z = ((c >> 16) & 0xff) * 0.003921568627f; //  /255.0f;
	rgba.w = ((c >> 24) & 0xff) * 0.003921568627f; //  /255.0f;
	return rgba;
}
//column pass using coalesced global memory reads
__global__ void
d_bilateral_filter(USHORT *od, int w, int h, int r)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	float sum = 0.0f;
	float factor = 0.0f;;
	uchar4 t = { 0, 0, 0, 0 };
	float tw = 0.f, tx = 0.f, ty = 0.f, tz = 0.f;
	uchar4 center = tex2D(rgbaTex, x, y);
	//t = center;
	int posIndex = 0;
	for (int i = -r; i <= r; i++)
	{
		for (int j = -r; j <= r; j++)
		{
			uchar4 curPix = { 0, 0, 0, 0 };
			USHORT d = (USHORT)sqrt((double)i*i + (double)j*j);
			if (d>r)
				continue;

			if (x + j<0 || y + i<0 || x + j>w - 1 || y + i>h - 1)
			{
				factor = 0;
			}
			else
			{
				curPix = tex2D(rgbaTex, x + j, y + i);
				factor = space_weight[d] *     //domain factor
					colorLenGaussian(curPix, center);             //range factor
			}


			tw += factor * (float)curPix.w;
			tx += factor * (float)curPix.x;
			ty += factor * (float)curPix.y;
			tz += factor * (float)curPix.z;
			sum += factor;
		}
	}
	t.w = (UCHAR)(tw / sum);
	t.x = (UCHAR)(tx / sum);
	t.y = (UCHAR)(ty / sum);
	t.z = (UCHAR)(tz / sum);
	od[y * w + x] = (USHORT)(((UINT)t.w) << 24 | ((UINT)t.z) << 16 | ((UINT)t.y) << 8 | ((UINT)t.x));

}

extern "C"
void updateGaussian(float sigma_color, float sigma_space, int radius)
{
	if (sigma_color <= 0)
		sigma_color = 1;
	if (sigma_space <= 0)
		sigma_space = 1;
	double gauss_color_coeff = -0.5 / (sigma_color*sigma_color);
	double gauss_space_coeff = -0.5 / (sigma_space*sigma_space);

	float color_gaussian[4 * 256];
	float space_gaussian[1024];

	for (int i = 0; i<256 * 4; i++)
	{
		color_gaussian[i] = (float)std::exp(i*i*gauss_color_coeff);
		space_gaussian[i] = (float)std::exp(i*i*gauss_space_coeff);
		//if(i>100) color_gaussian[i] = 0.0f; //����Ч��
	}
	// 	for(int i = -radius,int maxk=0;i<radius;i++)
	// 		for(int j=-radius;j<radius;j++)
	// 		{
	// 			double r = sqrt((double)i*i + (double)j*j);
	// 			 if( r > radius )
	//                 continue;  
	// 			space_gaussian[maxk++] = (float)std::exp(r*r*gauss_space_coeff); 
	// 			//space_ofs[maxk++] = (int)(i*temp.step + j*4);  
	// 		}

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(color_weight), color_gaussian, sizeof(float)*(4 * 256)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(space_weight), space_gaussian, sizeof(float)*(1024)));
}

//---------------------��ʼ���ڴ�---------------------
extern "C"
void initTexture(int width, int height, USHORT *hImage)
{
	// copy image data to array
	//  hipMallocPitch�ǽ��ڴ氲װ����ķ�ʽ���п����ڴ棬����ʹ�õ�ʱ����Ҫ����pitchֵ����ʾ��ǰ�洢ÿ����pitch�У�����ʱʹ��a[pitch*i+j]����ԭ����a[row*i+j]
	checkCudaErrors(hipMallocPitch(&dImage, &pitch, sizeof(USHORT)*width, height));
	checkCudaErrors(hipMallocPitch(&dTemp, &pitch, sizeof(USHORT)*width, height));
	checkCudaErrors(hipMemcpy2D(dImage, pitch, hImage, sizeof(USHORT)*width,
		sizeof(USHORT)*width, height, hipMemcpyHostToDevice));
}

//---------------------�ͷ��ڴ�----------------------------
extern "C"
void freeTextures()
{
	checkCudaErrors(hipFree(dImage));
	checkCudaErrors(hipFree(dTemp));
}

// RGBA version
extern "C"
double bilateralFilterRGBA(USHORT *dDest,
	int width, int height,
	int radius, int iterations,
	StopWatchInterface *timer)
{
	// var for kernel computation timing
	double dKernelTime;

	// Bind the array to the texture
	hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
	checkCudaErrors(hipBindTexture2D(0, rgbaTex, dImage, desc, width, height, pitch));

	for (int i = 0; i<iterations; i++)
	{
		// sync host and start kernel computation timer
		dKernelTime = 0.0;
		checkCudaErrors(hipDeviceSynchronize());
		sdkResetTimer(&timer);

		dim3 blockSize(16, 16);
		dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);

		d_bilateral_filter << < gridSize, blockSize >> >(dDest, width, height, radius);

		// sync host and stop computation timer
		checkCudaErrors(hipDeviceSynchronize());
		dKernelTime += sdkGetTimerValue(&timer);

	}

	return ((dKernelTime / 1000.) / (double)iterations);
}

//-----------------------GPU�ӿ�ʵ��--------------------
//GPU_Bilatetal_Filter()
//{
//
//}