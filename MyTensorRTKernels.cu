#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <fstream>
#include <hip/hip_runtime_api.h>
#include <memory>
#include <string>
#include <iostream>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>

__global__ void hwcToNchwKernel(const float* src, float* dst, 
                                int height, int width, int channels,
                                float mean0, float mean1, float mean2,
                                float std0, float std1, float std2) 
{
    // ����ȫ������
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPixels = height * width;
    
    if (idx < totalPixels) {
        // ��������λ��
        const int y = idx / width;
        const int x = idx % width;
        
        // ԭʼ�������� (HWC����)
        const int hwcIdx = (y * width + x) * channels;
        
        // ����ÿ��ͨ��
        for (int c = 0; c < channels; c++) {
            // NCHW��������
            const int nchwIdx = c * totalPixels + idx;
            
            // Ӧ�ñ�׼��
            float value = src[hwcIdx + c];
            
            // Ӧ��ͨ���ض��ı�׼��
            float normalized;
            if (c == 0) {
                normalized = (value - mean0) / std0;
            } else if (c == 1) {
                normalized = (value - mean1) / std1;
            } else {
                normalized = (value - mean2) / std2;
            }
            
            // ת��ΪFP16
            dst[nchwIdx] = (normalized);
        }
    }
}

void convertHWCtoNCHW(cv::cuda::GpuMat& hwcInput, float* nchwOutput, 
                     hipStream_t& stream,
                     const float mean[3], const float std[3])
{
    const int height = hwcInput.rows;
    const int width = hwcInput.cols;
    const int channels = hwcInput.channels();
    const int totalPixels = height * width;
    
    // �����߳̿�������С
    const int blockSize = 256;
    const int gridSize = (totalPixels + blockSize - 1) / blockSize;
    
    // ���ú˺���
    hwcToNchwKernel<<<gridSize, blockSize, 0, stream>>>(
        reinterpret_cast<const float*>(hwcInput.data), 
        nchwOutput, 
        height, 
        width, 
        channels,
        mean[0], mean[1], mean[2],
        std[0], std[1], std[2]
    );
    
    // ������
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel error: " + std::string(hipGetErrorString(err)));
    }
}
__global__ void convert_HWC_to_NCHW_and_normalize_kernel(
    const float* src,
    size_t src_step,
    float* dst,
    int width,
    int height,
    float mean_r, float mean_g, float mean_b,
    float std_r, float std_g, float std_b)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    // 1. ��ȷ����Դ��ַ (HWC)
    // src_step �����ֽ�Ϊ��λ�ģ�����Ҫ��ת�� char*
    const float* p_src_pixel = (const float*)((const char*)src + y * src_step) + x * 3;

    // 2. ��ȡ RGB ֵ (���������Ѿ���RGB)
    float r = p_src_pixel[0];
    float g = p_src_pixel[1];
    float b = p_src_pixel[2];

    // 3. �ڼĴ�������ɱ�׼��
    r = (r - mean_r) / std_r;
    g = (g - mean_g) / std_g;
    b = (b - mean_b) / std_b;

    // 4. ֱ��д�뵽Ŀ�� NCHW ��ʽ����ȷλ��
    size_t channel_offset = (size_t)height * width;
    dst[y * width + x] = r;                       // д�� R ͨ��ƽ��
    dst[channel_offset + y * width + x] = g;      // д�� G ͨ��ƽ��
    dst[2 * channel_offset + y * width + x] = b;  // д�� B ͨ��ƽ��
}

// C++ ��װ������ʵ��
void convert_HWC_to_NCHW_and_normalize(
    const cv::cuda::GpuMat& src, 
    float* dst, 
    int width, 
    int height, 
    const float* mean,
    const float* std,
    hipStream_t stream)
{
    dim3 block(16, 16); // �߳̿��С
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    convert_HWC_to_NCHW_and_normalize_kernel<<<grid, block, 0, stream>>>(
        (const float*)src.data,
        src.step,
        dst,
        width,
        height,
        mean[0], mean[1], mean[2], // ��mean/stdֱ����Ϊֵ���ݣ�Ч�ʸ���
        std[0], std[1], std[2]
    );
}