#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <fstream>
#include <hip/hip_runtime_api.h>
#include <memory>
#include <string>
#include <iostream>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>

__global__ void hwcToNchwKernel(const float* src, float* dst, 
                                int height, int width, int channels,
                                float mean0, float mean1, float mean2,
                                float std0, float std1, float std2) 
{
    // ����ȫ������
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPixels = height * width;
    
    if (idx < totalPixels) {
        // ��������λ��
        const int y = idx / width;
        const int x = idx % width;
        
        // ԭʼ�������� (HWC����)
        const int hwcIdx = (y * width + x) * channels;
        
        // ����ÿ��ͨ��
        for (int c = 0; c < channels; c++) {
            // NCHW��������
            const int nchwIdx = c * totalPixels + idx;
            
            // Ӧ�ñ�׼��
            float value = src[hwcIdx + c];
            
            // Ӧ��ͨ���ض��ı�׼��
            float normalized;
            if (c == 0) {
                normalized = (value - mean0) / std0;
            } else if (c == 1) {
                normalized = (value - mean1) / std1;
            } else {
                normalized = (value - mean2) / std2;
            }
            
            // ת��ΪFP16
            dst[nchwIdx] = (normalized);
        }
    }
}

void convertHWCtoNCHW(cv::cuda::GpuMat& hwcInput, float* nchwOutput, 
                     hipStream_t& stream,
                     const float mean[3], const float std[3])
{
    const int height = hwcInput.rows;
    const int width = hwcInput.cols;
    const int channels = hwcInput.channels();
    const int totalPixels = height * width;
    
    // �����߳̿�������С
    const int blockSize = 256;
    const int gridSize = (totalPixels + blockSize - 1) / blockSize;
    
    // ���ú˺���
    hwcToNchwKernel<<<gridSize, blockSize, 0, stream>>>(
        reinterpret_cast<const float*>(hwcInput.data), 
        nchwOutput, 
        height, 
        width, 
        channels,
        mean[0], mean[1], mean[2],
        std[0], std[1], std[2]
    );
    
    // ������
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA kernel error: " + std::string(hipGetErrorString(err)));
    }
}